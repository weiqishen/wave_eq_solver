#include "hip/hip_runtime.h"

//****************************************************************************

#include "reference_calc.cpp"
#include "utils.h"

__global__
void solver(const unsigned char* const inputsolverspace,
                   unsigned char* const outputsolverspace,
                   int numRows, int numCols,
                   const float* const filter, const int filterWidth)
{
 

  int px = blockIdx.x * blockDim.x + threadIdx.x;
  int py = blockIdx.y * blockDim.y + threadIdx.y;
  if (px >= numCols || py >= numRows) {
      return;
  }

  float c = 0.0f;

  for (int fx = 0; fx < filterWidth; fx++) {
    for (int fy = 0; fy < filterWidth; fy++) {
      int solverx = px + fx - filterWidth / 2;
      int solvery = py + fy - filterWidth / 2;
      solverx = min(max(solverx,0),numCols-1);
      solvery = min(max(solvery,0),numRows-1);
      c += (filter[fy*filterWidth+fx] * inputsolverspace[solvery*numCols+solverx]);
    }
  }

  outputsolverspace[py*numCols+px] = c;
}

//This kernel takes in an solver represented as a uchar4 and splits
//it into three solvers consisting of only one color solverspace each
__global__
void separatesolverspaces(const uchar4* const inputsolverRGBA,
                      int numRows,
                      int numCols,
                      unsigned char* const solver_tsolverspace,
                      unsigned char* const solver_t_2solverspace,
                      unsigned char* const solver_t_3solverspace)
{
  // TODO

  int px = blockIdx.x * blockDim.x + threadIdx.x;
  int py = blockIdx.y * blockDim.y + threadIdx.y;
  if (px >= numCols || py >= numRows) {
      return;
  }
  int i = py * numCols + px;
  solver_tsolverspace[i] = inputsolverRGBA[i].x;
  solver_t_2solverspace[i] = inputsolverRGBA[i].y;
  solver_t_3solverspace[i] = inputsolverRGBA[i].z;
}

__global__
void recombinesolverspaces(const unsigned char* const solver_tsolverspace,
                       const unsigned char* const solver_t_2solverspace,
                       const unsigned char* const solver_t_3solverspace,
                       uchar4* const outputsolverRGBA,
                       int numRows,
                       int numCols)
{
  const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);

  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

  //make sure we don't try and access memory outside the solver
  //by having any threads mapped there return early
  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    return;

  unsigned char solver_t   = solver_tsolverspace[thread_1D_pos];
  unsigned char solver_t_2 = solver_t_2solverspace[thread_1D_pos];
  unsigned char solver_t_3  = solver_t_3solverspace[thread_1D_pos];

  //Alpha should be 255 for no transparency
  uchar4 outputPixel = make_uchar4(solver_t, solver_t_2, solver_t_3, 255);

  outputsolverRGBA[thread_1D_pos] = outputPixel;
}

unsigned char *d_solver_t, *d_solver_t_2, *d_solver_t_3;
float         *d_filter;

void allocateMemoryAndCopyToGPU(const size_t numRowssolver, const size_t numColssolver,
                                const float* const h_filter, const size_t filterWidth)
{

  //allocate memory for the three different solverspaces
  checkCudaErrors(hipMalloc(&d_solver_t,   sizeof(unsigned char) * numRowssolver * numColssolver));
  checkCudaErrors(hipMalloc(&d_solver_t_2, sizeof(unsigned char) * numRowssolver * numColssolver));
  checkCudaErrors(hipMalloc(&d_solver_t_3,  sizeof(unsigned char) * numRowssolver * numColssolver));

  //TODO:
  //Allocate memory for the filter on the GPU
  //Use the pointer d_filter that we have already declasolver_t for you
  //You need to allocate memory for the filter with hipMalloc
  //be sure to use checkCudaErrors like the above examples to
  //be able to tell if anything goes wrong
  //IMPORTANT: Notice that we pass a pointer to a pointer to hipMalloc
  checkCudaErrors(hipMalloc(&d_filter, sizeof(float) * filterWidth * filterWidth));

  //TODO:
  //Copy the filter on the host (h_filter) to the memory you just allocated
  //on the GPU.  hipMemcpy(dst, src, numBytes, hipMemcpyHostToDevice);
  //Remember to use checkCudaErrors!
  checkCudaErrors(hipMemcpy(d_filter, h_filter, sizeof(float) * filterWidth * filterWidth, hipMemcpyHostToDevice));
}

void your_kernel_(const uchar4 * const h_inputsolverRGBA, uchar4 * const d_inputsolverRGBA,
                        uchar4* const d_outputsolverRGBA, const size_t numRows, const size_t numCols,
                        unsigned char *d_solver_tsolver_t, 
                        unsigned char *d_solver_t_2solver_t, 
                        unsigned char *d_solver_t_3solver_t,
                        const int filterWidth)
{
  //TODO: Set reasonable block size (i.e., number of threads per block)
  const dim3 blockSize(16,16,1);

  //TODO:
  //Compute correct grid size (i.e., number of blocks per kernel launch)
  //from the solver size and and block size.
  const dim3 gridSize(numCols/blockSize.x+1,numRows/blockSize.y+1,1);

  //TODO: Launch a kernel for separating the RGBA solver into different color solverspaces
  separatesolverspaces<<<gridSize, blockSize>>>(d_inputsolverRGBA,numRows,numCols,d_solver_t,d_solver_t_2,d_solver_t_3);

  // Call hipDeviceSynchronize(), then call checkCudaErrors() immediately after
  // launching your kernel to make sure that you didn't make any mistakes.
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  //TODO: Call your convolution kernel here 3 times, once for each color solverspace.
  kernel_<<<gridSize, blockSize>>>(d_solver_t,d_solver_tsolver_t,numRows,numCols,d_filter,filterWidth);
  kernel_<<<gridSize, blockSize>>>(d_solver_t_2,d_solver_t_2solver_t,numRows,numCols,d_filter,filterWidth);
  kernel_<<<gridSize, blockSize>>>(d_solver_t_3,d_solver_t_3solver_t,numRows,numCols,d_filter,filterWidth);

  // Again, call hipDeviceSynchronize(), then call checkCudaErrors() immediately after
  // launching your kernel to make sure that you didn't make any mistakes.
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  // Now we recombine your results. We take care of launching this kernel for you.
  //
  // NOTE: This kernel launch depends on the gridSize and blockSize variables,
  // which you must set yourself.
  recombinesolverspaces<<<gridSize, blockSize>>>(d_solver_tsolver_t,
                                             d_solver_t_2solver_t,
                                             d_solver_t_3solver_t,
                                             d_outputsolverRGBA,
                                             numRows,
                                             numCols);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}


//Free all the memory that we allocated
//TODO: make sure you free any arrays that you allocated
void cleanup() {
  checkCudaErrors(hipFree(d_solver_t));
  checkCudaErrors(hipFree(d_solver_t_2));
  checkCudaErrors(hipFree(d_solver_t_3));
}
